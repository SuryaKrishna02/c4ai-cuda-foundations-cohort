#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void kernel(float *d_odata, float *d_idata, int size_x, int size_y, int NUM_REPS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size_x * size_y)
    {
        for (int i = 0; i < NUM_REPS; ++i)
        {
            d_odata[idx] = d_idata[idx] + 1.0f;
        }
    }
}

int main()
{
    hipEvent_t start, stop;
    float time;

    // Error handling
    hipError_t err;

    // Create CUDA events
    err = hipEventCreate(&start);
    if (err != hipSuccess)
    {
        cerr << "Failed to create start event: " << hipGetErrorString(err) << endl;
        return -1;
    }

    err = hipEventCreate(&stop);
    if (err != hipSuccess)
    {
        cerr << "Failed to create stop event: " << hipGetErrorString(err) << endl;
        return -1;
    }

    // Example data size and kernel launch configuration
    int size_x = 1024;
    int size_y = 1024;
    int NUM_REPS = 10;
    int grid = (size_x * size_y + 255) / 256;
    int threads = 256;

    // Allocate device memory
    float *d_idata, *d_odata;
    err = hipMalloc((void **)&d_idata, size_x * size_y * sizeof(float));
    if (err != hipSuccess)
    {
        cerr << "Failed to allocate device memory for d_idata: " << hipGetErrorString(err) << endl;
        return -1;
    }

    err = hipMalloc((void **)&d_odata, size_x * size_y * sizeof(float));
    if (err != hipSuccess)
    {
        cerr << "Failed to allocate device memory for d_odata: " << hipGetErrorString(err) << endl;
        hipFree(d_idata);
        return -1;
    }

    // Record the start event
    err = hipEventRecord(start, 0);
    if (err != hipSuccess)
    {
        cerr << "Failed to record start event: " << hipGetErrorString(err) << endl;
        hipFree(d_idata);
        hipFree(d_odata);
        return -1;
    }

    // Launch the kernel
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, NUM_REPS);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        cerr << "Failed to launch kernel: " << hipGetErrorString(err) << endl;
        hipFree(d_idata);
        hipFree(d_odata);
        return -1;
    }

    // Record the stop event
    err = hipEventRecord(stop, 0);
    if (err != hipSuccess)
    {
        cerr << "Failed to record stop event: " << hipGetErrorString(err) << endl;
        hipFree(d_idata);
        hipFree(d_odata);
        return -1;
    }

    // Synchronize the stop event
    err = hipEventSynchronize(stop);
    if (err != hipSuccess)
    {
        cerr << "Failed to synchronize stop event: " << hipGetErrorString(err) << endl;
        hipFree(d_idata);
        hipFree(d_odata);
        return -1;
    }

    // Calculate the elapsed time
    err = hipEventElapsedTime(&time, start, stop);
    if (err != hipSuccess)
    {
        cerr << "Failed to calculate elapsed time: " << hipGetErrorString(err) << endl;
        hipFree(d_idata);
        hipFree(d_odata);
        return -1;
    }

    cout << "Elapsed time: " << time << " ms" << endl;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_idata);
    hipFree(d_odata);

    return 0;
}