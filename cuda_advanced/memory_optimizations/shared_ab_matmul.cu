#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_DIM 32 // Define TILE_DIM as the warp size, typically 32 for current GPUs

__global__ void coalescedMultiply(float *a, float *b, float *c, int M,
                                  int N)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM],
        bTile[TILE_DIM][TILE_DIM];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row * TILE_DIM + threadIdx.x];
    bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y * N + col];
    __syncthreads();
    if (row < M && col < N)
    {
        for (int i = 0; i < TILE_DIM; i++)
        {
            sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
        }
        c[row * N + col] = sum;
    }
}

int main()
{
    // Define matrix dimensions
    int M = 1024; // Example size for matrix A (Mxw)
    int w = 32;   // Warp size, also width of matrix A and height of matrix B
    int N = 1024; // Example size for matrix B (wxN)

    // Allocate host memory
    float *h_a = (float *)malloc(M * w * sizeof(float));
    float *h_b = (float *)malloc(w * N * sizeof(float));
    float *h_c = (float *)malloc(M * N * sizeof(float));

    // Initialize host matrices
    for (int i = 0; i < M * w; i++)
    {
        h_a[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < w * N; i++)
    {
        h_b[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, M * w * sizeof(float));
    hipMalloc(&d_b, w * N * sizeof(float));
    hipMalloc(&d_c, M * N * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_a, h_a, M * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, w * N * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    dim3 dimGrid((N + TILE_DIM - 1) / TILE_DIM, (M + TILE_DIM - 1) / TILE_DIM);

    // Launch kernel
    coalescedMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, M, N);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Check the results
    // (Checking code here)

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
