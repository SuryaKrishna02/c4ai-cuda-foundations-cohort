#include <hip/hip_runtime.h>
#include <iostream>

// Error checking macro
#define cudaCheckError()                                                                                                  \
    {                                                                                                                     \
        hipError_t e = hipGetLastError();                                                                               \
        if (e != hipSuccess)                                                                                             \
        {                                                                                                                 \
            std::cerr << "CUDA Error " << __FILE__ << " line " << __LINE__ << ": " << hipGetErrorString(e) << std::endl; \
            exit(EXIT_FAILURE);                                                                                           \
        }                                                                                                                 \
    }

__global__ void kernel(float *data)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    data[idx] *= 2.0f; // operation: double the value
}

int main()
{
    float *a_h, *a_d;
    float *otherData_d;
    size_t size = 1024 * sizeof(float); // Example size
    dim3 grid(1);
    dim3 block(1024);

    // Allocate host memory
    a_h = (float *)malloc(size);
    if (!a_h)
    {
        std::cerr << "Failed to allocate host memory" << std::endl;
        return EXIT_FAILURE;
    }

    // Allocate device memory
    hipMalloc((void **)&a_d, size);
    cudaCheckError();
    hipMalloc((void **)&otherData_d, size);
    cudaCheckError();

    // Initialize host data
    for (int i = 0; i < 1024; ++i)
    {
        a_h[i] = static_cast<float>(i);
    }

    hipStream_t stream1, stream2;

    // Create streams
    hipStreamCreate(&stream1);
    cudaCheckError();
    hipStreamCreate(&stream2);
    cudaCheckError();

    // Asynchronously copy data to the device on stream1
    hipMemcpyAsync(a_d, a_h, size, hipMemcpyHostToDevice, stream1);
    cudaCheckError();

    // Launch kernel on stream2
    kernel<<<grid, block, 0, stream2>>>(otherData_d);
    cudaCheckError();

    // Synchronize streams to ensure all operations are complete
    hipStreamSynchronize(stream1);
    cudaCheckError();
    hipStreamSynchronize(stream2);
    cudaCheckError();

    // Clean up
    hipStreamDestroy(stream1);
    cudaCheckError();
    hipStreamDestroy(stream2);
    cudaCheckError();
    hipFree(a_d);
    cudaCheckError();
    hipFree(otherData_d);
    cudaCheckError();
    free(a_h);

    return 0;
}