#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void initialize_array(int *array, int size)
{
    // Calculate the index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        array[index] = index;
    }
    // COMPLETE THIS
}

int main()
{
    const int array_size = 10;
    int *d_array;

    // Allocate memory on GPU
    hipMalloc((void **)&d_array, array_size * sizeof(int));

    // Launch the CUDA kernel to initialize the array
    initialize_array<<<1, array_size>>>(d_array, array_size);

    // Copy data from device to host
    int h_array[array_size];
    hipMemcpy(h_array, d_array, array_size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the initialized array
    cout << "Initialized Array:" << endl;
    for (int i = 0; i < array_size; ++i)
    {
        cout << h_array[i] << " ";
    }
    cout << endl;

    // Free GPU memory
    hipFree(d_array);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    return 0;
}