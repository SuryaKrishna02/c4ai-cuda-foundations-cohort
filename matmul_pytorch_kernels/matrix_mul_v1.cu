
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
using namespace std;

__global__ void matrixMul(int *a, int *b, int *c, int n)
{
    // Compute each thread's row
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Compute each thread's col
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;
    // Boundary protection
    if (row < n && col < n)
    {
        // Iterate over row, and down column
        for (int k = 0; k < n; k++)
        {
            // Accumulate result for a single element
            temp_sum += a[row * n + k] * b[k * n + col];
        }

        // Assign result
        c[row * n + col] = temp_sum;
    }
}

// Check result
void verify_result(int *a, int *b, int *c, int n)
{
    int *verify_c;
    verify_c = (int *)malloc(n * n * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            for (int k = 0; k < n; k++)
            {
                verify_c[i * n + j] += a[i * n + k] * b[k * n + j];
            }
        }
    }

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            assert(c[i * n + j] == verify_c[i * n + j]);
        }
    }
}

// Initialization function for matrices;
void init_matrices(int *a, int *b, int n)
{
    for (int i = 0; i < n * n; i++)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }
}

int main()
{
    // Matrix size of 1024 x 1024;
    int n = 1 << 10;

    // Size (in bytes) of matrix
    size_t bytes = n * n * sizeof(int);

    // Host pointers
    int *h_a, *h_b, *h_c;

    // Allocate host memory
    h_a = (int *)malloc(bytes);
    h_b = (int *)malloc(bytes);
    h_c = (int *)malloc(bytes);

    // Device pointers
    int *d_a, *d_b, *d_c;

    // Allocated device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Initialize matrices
    init_matrices(h_a, h_b, n);

    // Copy data to the device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Threads per block
    int BLOCK_SIZE = 16;

    // Blocks in each dimension
    int GRID_SIZE = (int)ceil(n / BLOCK_SIZE);

    // use dim3 objects
    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    // Launch Kernel
    matrixMul<<<grid, threads>>>(d_a, d_b, d_c, n);

    // Wait for GPU to complete execution
    hipDeviceSynchronize();

    // Copy back to the host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Check the result
    verify_result(h_a, h_b, h_c, n);

    // Free the GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free the CPU memory
    free(h_a);
    free(h_b);
    free(h_c);

    cout << "COMPLETED SUCCESSFULLY\n"
         << endl;

    return 0;
}